#include "hip/hip_runtime.h"
//----------------DeepPipe2 my implementation of Deep-Learning in Elixir-----------
//   I will port DeepPipe2 code from Elixir to Easy-ISLisp in 2021
/*


  
  
  
  __global__ void sigmoid_kernel(float *a, float *b, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {   
		  b[tid] = SIGMOID(a[tid]);
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  static ERL_NIF_TERM
  activate_sigmoid(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int n;
	  float *a,*b;
	  float *dev_a, *dev_b;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
  
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
  
		  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
  
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
  
	  sigmoid_kernel << <128, 128 >> >(dev_a, dev_b, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
	  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
  
	  return(b_bin);
  }
  
  
  
  __global__ void tanh_kernel(float *a, float *b, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {
		  b[tid] = tanh(a[tid]);
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  
  static ERL_NIF_TERM
  activate_tanh(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int n;
	  float *a,*b;
	  float *dev_a, *dev_b;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
  
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
  
		  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
  
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
  
	  tanh_kernel << <128, 128 >> >(dev_a, dev_b, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
	  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
  
	  return(b_bin);
  }
  
  
  
  __global__ void relu_kernel(float *a, float *b, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {   
		  if(a[tid] >= 0)
			  b[tid] = a[tid];
		  else 
			  b[tid] = 0.0;
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  
  static ERL_NIF_TERM
  activate_relu(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int n;
	  float *a,*b;
	  float *dev_a, *dev_b;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
  
		  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
  
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
  
	  relu_kernel << <128, 128 >> >(dev_a, dev_b, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
	  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
  
	  return(b_bin);
  }
  
  static ERL_NIF_TERM
  activate_softmax(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int r1, c1, n, i, j, k;
	  float *a,*b;
	  float max,sum,delta;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  n = r1*c1;
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
  
	  //calculate softmax
	  delta = 0.01;
	  for(i=0;i<r1;i++){
		  for(j=0;j<c1;j++){
			  max = -3.402823e38;
			  for(k=0;k<c1;k++){
				  if(a[IDX2C(i,k,r1)] > max)
					  max = a[IDX2C(i,k,r1)];
			  }
			  sum = 0.0;
			  for(k=0;k<c1;k++){
				  sum = sum + exp(a[IDX2C(i,k,r1)] - max);
			  }
			  b[IDX2C(i,j,r1)] = exp(a[IDX2C(i,j,r1)] - max) / (sum+delta);
			  
		  }
	  }
  
  
	  return(b_bin);
  }
  
  
  
  __global__ void differ_sigmoid_kernel(float *a, float *b, float *c, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {   
		  
		  c[tid] = a[tid] * ((1 - SIGMOID(b[tid])) * SIGMOID(b[tid]));
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  static ERL_NIF_TERM
  differ_sigmoid(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin, b_bin;
	  ERL_NIF_TERM  c_bin;
	  int n;
	  float *a,*b,*c;
	  float *dev_a, *dev_b, *dev_c;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);
  
		  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
  
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
  
	  differ_sigmoid_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
  
	  return(c_bin);
  }
  
  
  __global__ void differ_tanh_kernel(float *a, float *b, float *c, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {   
		  c[tid] = a[tid] * (1/(cosh(b[tid]) * cosh(b[tid])));
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  static ERL_NIF_TERM
  differ_tanh(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin, b_bin;
	  ERL_NIF_TERM  c_bin;
	  int n;
	  float *a,*b,*c;
	  float *dev_a, *dev_b, *dev_c;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
	  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);
  
		  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
  
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
  
	  differ_tanh_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
  
	  return(c_bin);
  }
  
  
  
  __global__ void differ_relu_kernel(float *a, float *b, float *c, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {   
		  if(b[tid] >= 0)
			  c[tid] = a[tid];
		  else 
			  c[tid] = 0.0;
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  static ERL_NIF_TERM
  differ_relu(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin, b_bin;
	  ERL_NIF_TERM  c_bin;
	  int n;
	  float *a,*b,*c;
	  float *dev_a, *dev_b, *dev_c;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,4);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);
  
		  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
  
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
  
	  differ_relu_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
  
	  return(c_bin);
  }
  
  
  __global__ void smult_kernel(float d, float *a, float *b, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {
		  b[tid] = d * a[tid];
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  
  static ERL_NIF_TERM
  smult1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int n;
	  float *a,*b;
	  float *dev_a, *dev_b;
	  double s;
  
	  if (!enif_get_double(env, argv[0], &s)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &n)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
  
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
  
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
  
	  smult_kernel << <128, 128 >> >((float)s,dev_a, dev_b, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
  
	  return(b_bin);
  }
  
  
  static ERL_NIF_TERM
  trace1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  result;
	  int r1, c1, i, j;
	  float *a;
	  float trace;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  a = (float *) a_bin.data;
	  
	  trace = 0.0;
	  for(i=0;i<r1;i++){
		  for(j=0;j<c1;j++){
			  if(i==j)
				  trace = trace + a[IDX2C(i,j,r1)];
		  }
	  }
  
	  result = enif_make_double(env,trace);
  
	  return(result);
  }
  
  
  static ERL_NIF_TERM
  mean_square(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  ERL_NIF_TERM  result;
	  int r1, c1, i, j;
	  float *a, *b;
	  float d,s;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &b_bin )) return enif_make_int(env,4);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
  
	  s = 0.0;
	  for(i=0;i<r1;i++){
		  for (j=0;j<c1;j++){
			  d = a[IDX2C(i,j,r1)] -  b[IDX2C(i,j,r1)];
			  s = s + d*d;            
		  }
	  } 
	  s = s / (2.0*(float(r1)));
	  result = enif_make_double(env,s);
	  return(result);
  }
  
  static ERL_NIF_TERM
  cross_entropy(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  ERL_NIF_TERM  result;
	  int r1, c1, i, j;
	  float *a, *b;
	  float d,s,delta;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &b_bin )) return enif_make_int(env,4);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
  
	  
	  delta = 1e-7;
	  s = 0.0;
	  for(i=0;i<r1;i++){
		  for (j=0;j<c1;j++){
			  d = a[IDX2C(i,j,r1)] + delta;
			  s = s + b[IDX2C(i,j,r1)] * log(d);
		  }
	  }
	  s = -1.0 * s / (float)r1;
	  result = enif_make_double(env,s);
	  return(result);
  }
  
  
  
  
  
  static ERL_NIF_TERM
  elt1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  result;
	  int r1, c1, i, j;
	  float *a;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &i)) enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &j)) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
	  a = (float *) a_bin.data;
	  
	  result = enif_make_double(env,(double)a[IDX2C(i,j,r1)]);
  
	  return(result);
  }
  
  static ERL_NIF_TERM
  set1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int r1, c1, n, i, j, x, y;
	  float *a,*b;
	  double val;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &x)) return enif_make_int(env,4);
	  if (!enif_get_int(env, argv[4], &y)) return enif_make_int(env,5);
	  if (!enif_get_double(env, argv[5], &val)) return enif_make_int(env,6);
  
  
	  n = r1*c1;
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
  
	  for(i=0;i<r1;i++){
		  for(j=0;j<c1;j++){
			  if(i==x && j==y)
				  b[IDX2C(i,j,r1)] = (float)val;
			  else 
				  b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)];
		  }
	  }
  
  
	  return(b_bin);
  }
  
  static ERL_NIF_TERM
  add_diff1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int r1, c1, n, i, j, x, y;
	  float *a,*b;
	  double val;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &x)) return enif_make_int(env,4);
	  if (!enif_get_int(env, argv[4], &y)) return enif_make_int(env,5);
	  if (!enif_get_double(env, argv[5], &val)) return enif_make_int(env,6);
  
  
	  n = r1*c1;
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
  
	  for(i=0;i<r1;i++){
		  for(j=0;j<c1;j++){
			  if(i==x && j==y)
				  b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)] + (float)val;
			  else 
				  b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)];
		  }
	  }
  
  
	  return(b_bin);
  }
  
  static ERL_NIF_TERM
  add_diff2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int n1, c1, h1, w1, n, i, j, k, l, n2, c2, h2, w2;
	  float *a,*b;
	  double val;
  
	  if (!enif_get_int(env, argv[0], &n1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &h1)) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &w1)) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
	  if (!enif_get_int(env, argv[5], &n2)) return enif_make_int(env,6);
	  if (!enif_get_int(env, argv[6], &c2)) return enif_make_int(env,7);
	  if (!enif_get_int(env, argv[7], &h2)) return enif_make_int(env,8);
	  if (!enif_get_int(env, argv[8], &w2)) return enif_make_int(env,9);
	  if (!enif_get_double(env, argv[9], &val)) return enif_make_int(env,10);
  
  
	  n = n1*c1*h1*w1;
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
  
	  
	  for(i=0;i<n1;i++){
		  for(j=0;j<c1;j++){
			  for(k=0;k<h1;k++){
				  for(l=0;l<w1;l++){
					  if(i==n2 && j==c2 && k==h2 && l==w2){
						  b[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(i,j,k,l,c1,h1,w1)] + (float)val;
					  }
					  else {
						  b[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(i,j,k,l,c1,h1,w1)];
					  }
				  }
			  }
		  }
	  }
  
  
	  return(b_bin);
  }
  
  
  
  static ERL_NIF_TERM
  average1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int r1, c1, i, j;
	  float *a,*b;
	  float sum;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, c1 * sizeof(float), &b_bin);
  
	  for(j=0;j<c1;j++){
		  sum = 0.0;
		  for(i=0;i<r1;i++){
			  sum = sum + a[IDX2C(i,j,r1)];
		  }
		  b[j] = sum / (float)r1;
	  }
  
  
	  return(b_bin);
  }
  
  /*
  1st arg row-size of matrix
  2nd arg col-size of matrix
  3rd arg matrix data binary
  */
  
  
  static ERL_NIF_TERM
  sum1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  result;
	  int r1, c1, i, j;
	  float *a;
	  float sum;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  a = (float *) a_bin.data;
	  
	  sum = 0.0;
	  for(i=0;i<r1;i++){
		  for(j=0;j<c1;j++){
			  sum = sum + a[IDX2C(i,j,r1)];
		  }
	  }
  
	  result = enif_make_double(env,sum);
  
	  return(result);
  }
  
  /*
  transfer 2 DIm matrix to list 
  */
  static ERL_NIF_TERM
  to_list1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  head,list;
	  int r1, c1, i, j;
	  float *a;
  
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  a = (float *) a_bin.data;
  
	  
	  list = enif_make_list(env, 0);
	  for(i=r1-1;i>=0;i--){
		  for(j=c1-1;j>=0;j--){
			  head = enif_make_double(env,(double)a[IDX2C(i,j,r1)]);
			  list = enif_make_list_cell(env,head,list);
		  }
	  }
  
	  return(list);
  }
  /*
  transfer 3 DIm matrix to list
  */
  
  static ERL_NIF_TERM
  to_list2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  head,list;
	  int c, h, w, i, j, k;
	  float *a;
  
	  if (!enif_get_int(env, argv[0], &c)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &h)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &w)) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &a_bin )) return enif_make_int(env,4);
	 
	  a = (float *) a_bin.data;
	  
	  list = enif_make_list(env, 0);
	  for(i=c-1;i>=0;i--){
		  for(j=h-1;j>=0;j--){
			  for(k=w-1;k>=0;k--){
				  head = enif_make_double(env,(double)a[IDX3C(i,j,k,h,w)]);
				  list = enif_make_list_cell(env,head,list);
			  }
		  }
	  }
  
	  return(list);
  }
  /*
  transfer 4 DIm matrix to list
  */
  static ERL_NIF_TERM
  to_list3(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  head,list;
	  int n, c, h, w, i, j, k, l;
	  float *a;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &h)) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &w)) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_badarg(env);
	  a = (float *) a_bin.data;
  
	  
	  list = enif_make_list(env, 0);
	  for(i=n-1;i>=0;i--){
		  for(j=c-1;j>=0;j--){
			  for(k=h-1;k>=0;k--){
				  for(l=w-1;l>=0;l--){
					  head = enif_make_double(env,(double)a[IDX4C(i,j,k,l,c,h,w)]);
					  list = enif_make_list_cell(env,head,list);
				  }
			  }
		  }
	  }
  
	  return(list);
  }
  
  __global__ void dropout1_kernel(float *a, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {
		  a[tid] = 1.0;
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  /*
  1st arg size of mask tensor
  2nd arg rate of dropout
  
  return mask tensor
  element of mask tensor is basicaly 1.0.
  element of dropout rate is 0.0.
  when forward and backward, generate Hadamard product with mask tensor
  */
  static ERL_NIF_TERM
  dropout1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ERL_NIF_TERM  a_bin;
	  int n,count,i,j;
	  float *a,*dev_a;
	  double dropout_rate;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_get_double(env, argv[1], &dropout_rate)) return enif_make_int(env,2);
  
	  a = (float *) enif_make_new_binary(env, n * sizeof(float), &a_bin);
  
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
  
	  dropout1_kernel << <128, 128 >> >(dev_a, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(a, dev_a, n * sizeof(float), hipMemcpyDeviceToHost));
  
  
	  // dropout
	  count = (int)(double(n)*dropout_rate);
	  for(i=0;i<count;i++){
		  j = rand() % n;
		  a[j] = 0.0;
	  }
  
	  // free 
	  hipFree(dev_a);
  
	  return(a_bin);
  }
  
  
  __global__ void sgd1_kernel(float *a, float *b, float *c, float lr, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {
		  c[tid] = a[tid] - b[tid]*lr;
		  tid += blockDim.x * gridDim.x;
	  }
  }
  /*
  w - g*lr
  w is weight matrix.
  g is gradient matrix.
  when element of w is zero result is zero. This means dropout.
  return updated weight matrix.
  
  1st arg is size of vectorized matrix
  2nd arg is weight matrix or tensor
  3rd arg is gradient matrix or tensor
  4th arg is learning rate
  */
  static ERL_NIF_TERM
  sgd1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin, b_bin;
	  ERL_NIF_TERM  c_bin;
	  int n;
	  float *a,*b,*c,*dev_a, *dev_b, *dev_c;
	  float lr;
	  double learning_rate;
  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
	  if (!enif_get_double(env, argv[3], &learning_rate)) return enif_make_int(env,4);
  
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);
	  lr = (float) learning_rate;
  
		  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
  
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
  
	  sgd1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, lr, n);
  
	  // copy to host c from GPU dev_c
	  CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));
  
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);    
	  hipFree(dev_c);
  
	  return(c_bin);
  }
  
  
  /*
	def momentum(v, g, lr) do
	  Matrex.apply(v, g, fn v, g -> 0.5 * v - lr * g end)
	end
  */
  __global__ void momentum_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {   
		  
		  d[tid] = ((0.9 * b[tid]) - (lr * c[tid]));
		  e[tid] = a[tid] + d[tid];
		  
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  /*
  1st arg row-size of vectorized each-matrix
  2nd arg wight-matrix    (a)
  3rd arg v-matrix        (b)
  4th arg gradient-matrix (c)
  5th arg learning rate
  
  return tuple {next_v-mattrix,weight_matrix}
  */
  static ERL_NIF_TERM
  momentum1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin,c_bin;
	  ERL_NIF_TERM  d_bin,e_bin,tuple;
	  int n;
	  float *a,*b,*c,*d,*e;
	  float *dev_a, *dev_b, *dev_c ,*dev_d, *dev_e;
	  float lr;
	  double learning_rate;
	
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &c_bin )) return enif_make_int(env,4);
	  if (!enif_get_double(env, argv[4], &learning_rate)) return enif_make_int(env,5);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) c_bin.data;
	  d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
	  e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
	  lr = (float) learning_rate;
	  
	
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
	
	  momentum_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
	
	  // copy to host d from GPU dev_d
	  CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
	  hipFree(dev_d);
	  hipFree(dev_e);
	  
	  tuple = enif_make_tuple2(env,d_bin,e_bin);
	  return(tuple);
  }
  
  /* ADAGRAD
	  h1 = h + grad*grad
	  lr1 = lr/(sqrt(h1))
	  w1 = w - lr1 * grad 
  
	  a[] = w
	  b[] = h
	  c[] = grad
	  d[] = h1
	  e[] = w1
  */
	
  __global__ void adagrad_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  float lr1;
	  while (tid < n)
	  {   
		  d[tid] = b[tid] + c[tid]*c[tid];
		  if(d[tid] != 0.0)
			  lr1 = lr/(sqrt(d[tid]));
		  else
			  lr1 = lr;
		  e[tid] = a[tid] - lr1 * c[tid];
  
		  tid += blockDim.x * gridDim.x;
	  }
  }
   
  /*
  1st arg row-size of vectorized each-matrix
  2nd arg wight-matrix (a_bin)
  3rd arg h-matrix     (b_bin)
  4th arg grad-matrix  (c_bin)
  5th arg learning rate
  return tuple {new-h,new-w}
  */
  static ERL_NIF_TERM
  adagrad1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin,c_bin;
	  ERL_NIF_TERM  d_bin,e_bin,tuple;
	  int n;
	  float *a,*b,*c,*d,*e;
	  float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e;
	  float lr;
	  double learning_rate;
	  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &c_bin)) return enif_make_int(env,4);
	  if (!enif_get_double(env, argv[4], &learning_rate)) return enif_make_int(env,5);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) c_bin.data;
	  d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
	  e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
	  lr = (float) learning_rate;
	
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
	
	  adagrad_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
	
	  // copy to host d,e from GPU dev_d,dev_e
	  CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
	  hipFree(dev_d);
	  hipFree(dev_e);
	  
	  tuple = enif_make_tuple2(env,d_bin,e_bin);
	  return(tuple);
  }
  
  /* RMSprop
	  h1 = alpha * h + (1 - alpha) * grad*grad
	  lr1 = lr /(sqrt(h) + epsilon)
	  w1 = w - lr1 * grad 
  
	  a[] = w
	  b[] = h
	  c[] = grad
	  d[] = h1
	  e[] = w1
  */
	
  __global__ void rms_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  float lr1,alpha,epsilon;
	  alpha = 0.99;
	  epsilon = 10.0e-7;
	  while (tid < n)
	  {   
		  d[tid] = alpha * b[tid] + (1-alpha)*c[tid]*c[tid];
		  lr1 = lr/(sqrt(d[tid])+epsilon);
		  e[tid] = a[tid] - lr1*c[tid];
  
		  tid += blockDim.x * gridDim.x;
	  }
  }
   
  /*
  1st arg row-size of vectorized each-matrix
  2nd arg wight-matrix (a_bin)
  3rd arg h-matrix     (b_bin)
  4th arg grad-matrix  (c_bin)
  5th arg learning rate
  return tuple {new-h,new-w}
  */
  static ERL_NIF_TERM
  rms1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin,c_bin;
	  ERL_NIF_TERM  d_bin,e_bin,tuple;
	  int n;
	  float *a,*b,*c,*d,*e;
	  float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e;
	  float lr;
	  double learning_rate;
	  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &c_bin)) return enif_make_int(env,4);
	  if (!enif_get_double(env, argv[4], &learning_rate)) return enif_make_int(env,5);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) c_bin.data;
	  d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
	  e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
	  lr = (float) learning_rate;
	
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
	
	  rms_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
	
	  // copy to host d,e from GPU dev_d,dev_e
	  CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
	  hipFree(dev_d);
	  hipFree(dev_e);
	  
	  tuple = enif_make_tuple2(env,d_bin,e_bin);
	  return(tuple);
  }
  
  
  /* ADAM
	  beta1 = 0.9
	  beta2 = 0.999
	  epsilon = 10.0e-7
	  alpha = 0.001
	  m1 = beta1 * m + (1 - beta1) * grad
	  v1 = beta2 * v + (1 - beta2) * grad^2
	  m2 = m1/(1 - beta1)
	  v2 = v1/(1 - beta2)
	  w1 = w - alpha * m2/(sqrt(v2)+epsilon)
  
	  a[] is w
	  b[] is m
	  c[] is v
	  d[] is grad
	  e[] is m1
	  f[] is v1
	  g[] is w1
   */
	
  __global__ void adam_kernel(float *a, float *b, float *c, float *d, float *e, float *f, float *g, float lr, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  float beta1,beta2,epsilon,m2,v2;
	  beta1 = 0.9;
	  beta2 = 0.999;
	  epsilon = 10.0e-7;
	  //alpha = 0.001;
  
	  while (tid < n){   
		  e[tid] = beta1 * b[tid] + (1 - beta1) * d[tid];
		  f[tid] = beta2 * c[tid] + (1 - beta2) * d[tid]*d[tid];
		  m2 = e[tid]/(1-beta1);
		  v2 = f[tid]/(1-beta2);
		  g[tid] = a[tid] - lr * (m2/(sqrt(v2)+epsilon));
		   
		  tid += blockDim.x * gridDim.x;
	  }
  }
	 
  /*
  1st arg row-size of vectorized each-matrix
  2nd arg w-matrix     (a_bin)
  3rd arg m-matrix     (b_bin)
  4th arg v-matrix     (c_bin)
  5th arg grad-matrix  (d_bin)
  6th arg learning rate
  
  return tuple {m1,v1,w1}
  */
  static ERL_NIF_TERM
  adam1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin,c_bin,d_bin;
	  ERL_NIF_TERM  e_bin,f_bin,g_bin,tuple;
	  int n;
	  float *a,*b,*c,*d,*e,*f,*g;
	  float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e, *dev_f, *dev_g;
	  float lr;
	  double learning_rate;
		
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin)) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &c_bin)) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &d_bin)) return enif_make_int(env,5);
	  if (!enif_get_double(env, argv[5], &learning_rate)) return enif_make_int(env,6);
	
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) c_bin.data;
	  d = (float *) d_bin.data;
	  e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
	  f = (float *) enif_make_new_binary(env, n * sizeof(float), &f_bin);
	  g = (float *) enif_make_new_binary(env, n * sizeof(float), &g_bin);
	  lr = (float) learning_rate;
	  
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_f, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_g, n * sizeof(float)));
  
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_f, f, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_g, g, n * sizeof(float), hipMemcpyHostToDevice));
	  
	  adam_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, dev_f, dev_g, lr, n);
	  
	  // copy to host d,e from GPU dev_d,dev_e
	  CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(f, dev_f, n * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(g, dev_g, n * sizeof(float), hipMemcpyDeviceToHost));
		
	
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
	  hipFree(dev_d);
	  hipFree(dev_e);
	  hipFree(dev_f);
	  hipFree(dev_g);
	  
		
	  tuple = enif_make_tuple3(env,e_bin,f_bin,g_bin);
	  return(tuple);
  }
	
	
  
  
  /*
  1st arg row-size of matrix
  2nd arg col-size of matris
  3rd arg predicted matrix
  4th arg list of label. each element is integer
  return accuracy rate
  */
  
  static ERL_NIF_TERM
  accuracy1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  head,list,result;
	  int r1, c1, i, j, n, index,sum;
	  float *a;
	  double max,rate;
	
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  
  
	  // calculate accuracy
	  sum = 0;
	  list = argv[3]; 
	  for(i=0;i<r1;i++){
		  max = 0.0;
		  enif_get_list_cell(env, list, &head, &list);
		  enif_get_int(env,head,&n);
		  for(j=0;j<c1;j++){
			  if(a[IDX2C(i,j,r1)] > max){
				  max = a[IDX2C(i,j,r1)];
				  index = j;
			  }
		  }
		  if(index == n)
			  sum++;
	  }
	  rate = (double)sum / (double)r1;
	  result = enif_make_double(env,rate);
	  return(result);
  }
  
  /*
  1st arg row-size of matrix
  2nd arg col-size of matris
  3rd arg predicted matrix
  4th arg list of label. each element is integer
  return correct number
  */
  
  static ERL_NIF_TERM
  correct1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  head,list,result;
	  int r1, c1, i, j, n, index,sum;
	  float *a;
	  float max;
	
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  
  
	  // calculate correct number
	  sum = 0;
	  list = argv[3]; 
	  for(i=0;i<r1;i++){
		  max = 0.0;
		  enif_get_list_cell(env, list, &head, &list);
		  enif_get_int(env,head,&n);
		  for(j=0;j<c1;j++){
			  if(a[IDX2C(i,j,r1)] > max){
				  max = a[IDX2C(i,j,r1)];
				  index = j;
			  }
		  }
		  if(index == n)
			  sum++;
	  }
  
	  result = enif_make_double(env,(double)sum);
	  return(result);
  }
  
  
  /*
  random_select for matrix data
  */
  static ERL_NIF_TERM
  random_select1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  ERL_NIF_TERM  c_bin,d_bin,tuple;
	  int r1, c1, r2, c2, i, j, n, r;
	  float *a, *b, *c, *d;
	
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &r2)) return enif_make_int(env,4);
	  if (!enif_get_int(env, argv[4], &c2)) return enif_make_int(env,5);
	  if (!enif_inspect_binary(env, argv[5], &b_bin )) return enif_make_int(env,6);
	  if (!enif_get_int(env, argv[6], &n)) return enif_make_int(env,7);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n*c1 * sizeof(float), &c_bin);
	  d = (float *) enif_make_new_binary(env, n*c2 * sizeof(float), &d_bin);
  
  
	  // random-select
	  for(i=0;i<n;i++){
		  r = rand() % r1;
		  for(j=0;j<c1;j++){
			  c[IDX2C(i,j,n)] = a[IDX2C(r,j,r1)];
		  }
		  for(j=0;j<c2;j++){
			  d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
		  }    
	  }
  
	  tuple = enif_make_tuple2(env,c_bin,d_bin);
	  return(tuple);
  }
  
  /*
  random_select for 4D-tensor data
  */
  static ERL_NIF_TERM
  random_select2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  ERL_NIF_TERM  c_bin,d_bin,tuple;
	  int n1,c1,h1,w1,r2,c2, i, j, k, l, n, r;
	  float *a, *b, *c, *d;
	
	  if (!enif_get_int(env, argv[0], &n1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &h1)) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &w1)) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
	  if (!enif_get_int(env, argv[5], &r2)) return enif_make_int(env,6);
	  if (!enif_get_int(env, argv[6], &c2)) return enif_make_int(env,7);
	  if (!enif_inspect_binary(env, argv[7], &b_bin )) return enif_make_int(env,8);
	  if (!enif_get_int(env, argv[8], &n)) return enif_make_int(env,9);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n*c1*h1*w1 * sizeof(float), &c_bin);
	  d = (float *) enif_make_new_binary(env, n*r2*c2 * sizeof(float), &d_bin);
  
	  // random-select
	  for(i=0;i<n;i++){
		  r = rand() % n1;
		  for(j=0;j<c1;j++){
			  for(k=0;k<h1;k++){
				  for(l=0;l<w1;l++){
					  c[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(r,j,k,l,c1,h1,w1)];
				  }
			  }
		  }
		  for(j=0;j<c2;j++){
			  d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
		  }    
	  }
  
	  tuple = enif_make_tuple2(env,c_bin,d_bin);
	  return(tuple);
  }
  
  /*
  random_select for 3D-tensor data
  */
  static ERL_NIF_TERM
  random_select3(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  ERL_NIF_TERM  c_bin,d_bin,tuple;
	  int n1,h1,w1,r2,c2, i, j, k, n, r;
	  float *a, *b, *c, *d;
	
	  if (!enif_get_int(env, argv[0], &n1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &h1)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &w1)) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &a_bin )) return enif_make_int(env,4);
	  if (!enif_get_int(env, argv[4], &r2)) return enif_make_int(env,5);
	  if (!enif_get_int(env, argv[5], &c2)) return enif_make_int(env,6);
	  if (!enif_inspect_binary(env, argv[6], &b_bin )) return enif_make_int(env,7);
	  if (!enif_get_int(env, argv[7], &n)) return enif_make_int(env,8);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n*h1*w1 * sizeof(float), &c_bin);
	  d = (float *) enif_make_new_binary(env, n*r2*c2 * sizeof(float), &d_bin);
  
	  // random-select
	  for(i=0;i<n;i++){
		  r = rand() % n1;
		  for(j=0;j<h1;j++){
			  for(k=0;k<w1;k++){
				  c[IDX3C(i,j,k,h1,w1)] = a[IDX3C(r,j,k,h1,w1)];
			  }
		  }
		  for(j=0;j<c2;j++){
			  d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
		  }    
	  }
  
	  tuple = enif_make_tuple2(env,c_bin,d_bin);
	  return(tuple);
  }
  
  
  
  static ERL_NIF_TERM
  is_near1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  int i, n, sw;
	  float *a, *b;
	
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
  
	  // near check
	  sw = 0;
	  for(i=0;i<n;i++){
		 if(fabsf(a[i]) > fabsf(b[i])*1.15 || fabsf(a[i]) < fabsf(b[i])*0.85){
			  printf("%f %f \r\n", a[i], b[i]);
			  sw = 1;
		  }
	  }
	  if(sw == 0)
		  return enif_make_int(env,1); //true
	  else
		  return enif_make_int(env,0); //false
  }
  
  static ERL_NIF_TERM
  is_equal1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  int i, n;
	  float *a, *b;
	
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
  
	  // equal check
	  for(i=0;i<n;i++){
		 if(a[i] != b[i]){
			  return enif_make_int(env,0); //false
		  }
	  }
	  
	  return enif_make_int(env,1); //true
  }
  
  
  
  static ERL_NIF_TERM
  analizer1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  int i, n, id;
	  float *a;
	  float max,min,sum;
	
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &id)) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
  
	  // near check
	  for(i=0;i<n;i++){
		  if(isnan(a[i])){
			  return enif_make_int(env,9999);
		  }
		  if(isinf(a[i])){
			  return enif_make_int(env,9998);
		  }
	  }
  
	  //find max min avarage
	  max = -999999999;
	  min = 999999999;
	  sum = 0;
	  for(i=0;i<n;i++){
		  if(a[i] > max)
			  max = a[i];
		  
		  if(a[i] < min)
			  min = a[i];
		  
		  sum = sum+a[i];
  
	  }
	  printf("id max min average\r\n");
	  printf("%d %f %f %f \r\n", id, max, min, sum/(float)n);
  
	  return enif_make_int(env,1);
  }
  
  
  
	
	/*
	1st arg in_n of tensor
	2nd arg in_c of tensor
	3rd arg in_h of tensor
	4th arg in_w of tensor
	5th arg binary of tensor
	
	*/
  static ERL_NIF_TERM
  standardize1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int in_n,in_c,in_h,in_w,n1,i,c1,h1,w1,count;
	  float *a,*b;
	  float sum,average;
	
	  if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
  
	  n1 = in_n * in_c * in_h * in_w;
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n1 * sizeof(float), &b_bin);
	 
	  
	  for(i=0;i<in_n;i++){
		  sum = 0.0;
		  for(c1=0;c1<in_c;c1++){
			  for(h1=0;h1<in_h;h1++){
				  for(w1=0;w1<in_w;w1++){
					  sum = sum + a[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)];
				  }
			  }
		  }
		  count = in_c * in_h * in_w;
		  average = sum / (float)count;
		  for(c1=0;c1<in_c;c1++){
			  for(h1=0;h1<in_h;h1++){
				  for(w1=0;w1<in_w;w1++){
					  b[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)] = a[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)] - average;
				  }
			  }
		  }
	  }
	  
	  return(b_bin);
  }
  
  
	
  /*
  1st arg in_n of 3D tensor
  2rd arg in_r of 3D tensor
  3th arg in_c of 3D tensor
  4th arg binary of tensor
  5th arg nth in_r of 3D tensor
	
  */
  static ERL_NIF_TERM
  pickup1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int in_n,in_row,in_col,nth,n1,i,j;
	  float *a,*b;
	  
	  if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &in_row)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &in_col)) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &a_bin )) return enif_make_int(env,4);
	  if (!enif_get_int(env, argv[4], &nth)) return enif_make_int(env,5);
  
	  n1 = in_n * in_col;
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n1 * sizeof(float), &b_bin);
	   
		
	  for(i=0;i<in_n;i++){
		  for(j=0;j<in_col;j++){
			  b[IDX2C(i,j,in_n)] = a[IDX3C(i,nth,j,in_row,in_col)];
		  }
	  }
		
	  return(b_bin);
  }
	
  
	
  /*
  1st arg size of tensor or matrix
  2rd arg binary of tensor or matrix
	
  */
  static ERL_NIF_TERM
  copy1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int n,i;
	  float *a,*b;
	  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
  
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
	   
		
	  for(i=0;i<n;i++){
		  b[i] = a[i];
	  }
		
	  return(b_bin);
  }
  
  
  static ERL_NIF_TERM
  slice1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin,c_bin,d_bin,e_bin,tuple;
	  int in_r,in_c,in_c1,i,j,n,bias;
	  float *a,*b,*c,*d,*e;
	  
	  if (!enif_get_int(env, argv[0], &in_r)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
  
	  in_c1 = in_c / 4;
	  n = in_r * (in_c / 4);
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
	  c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);
	  d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
	  e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
	   
		
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c1;j++){
			  b[IDX2C(i,j,in_r)] = a[IDX2C(i,j,in_r)]; 
		  }
	  }
	  bias = in_c / 4;
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c1;j++){
			  c[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
		  }
	  }
	  bias = 2 * (in_c / 4);
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c1;j++){
			  d[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
		  }
	  }
	  bias = 3 * (in_c / 4);
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c1;j++){
			  e[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
		  }
	  }
		
	  tuple = enif_make_tuple4(env,b_bin,c_bin,d_bin,e_bin);
	  return(tuple);
  }
  
  static ERL_NIF_TERM
  unslice1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  b_bin,c_bin,d_bin,e_bin;
	  ERL_NIF_TERM  a_bin;
	  int in_r,in_c,i,j,n,bias;
	  float *a,*b,*c,*d,*e;
	  
	  if (!enif_get_int(env, argv[0], &in_r)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &c_bin )) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &d_bin )) return enif_make_int(env,5);
	  if (!enif_inspect_binary(env, argv[5], &e_bin )) return enif_make_int(env,6);
  
	  n = in_r * in_c * 4;
	  a = (float *) enif_make_new_binary(env, n * sizeof(float), &a_bin);
	  b = (float *) b_bin.data;
	  c = (float *) c_bin.data;
	  d = (float *) d_bin.data;
	  e = (float *) e_bin.data;
	  
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c;j++){
			  a[IDX2C(i,j,in_r)] = b[IDX2C(i,j,in_r)];
		  }
	  }
	  bias = in_c;
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c;j++){
			  a[IDX2C(i,j+bias,in_r)] = c[IDX2C(i,j,in_r)]; 
		  }
	  }
	  bias = 2 * in_c;
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c;j++){
			  a[IDX2C(i,j+bias,in_r)] = d[IDX2C(i,j,in_r)] ; 
		  }
	  }
	  bias = 3 * in_c;
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c;j++){
			  a[IDX2C(i,j+bias,in_r)] = e[IDX2C(i,j,in_r)]; 
		  }
	  }
	  return(a_bin);
  }
  
  
  
  


*/