#include "hip/hip_runtime.h"
//----------------DeepPipe2 my implementation of Deep-Learning in Elixir-----------
//   I will port DeepPipe2 code from Elixir to Easy-ISLisp in 2021
/*



  /*
  random_select for matrix data
  */
  static ERL_NIF_TERM
  random_select1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  ERL_NIF_TERM  c_bin,d_bin,tuple;
	  int r1, c1, r2, c2, i, j, n, r;
	  float *a, *b, *c, *d;
	
	  if (!enif_get_int(env, argv[0], &r1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &r2)) return enif_make_int(env,4);
	  if (!enif_get_int(env, argv[4], &c2)) return enif_make_int(env,5);
	  if (!enif_inspect_binary(env, argv[5], &b_bin )) return enif_make_int(env,6);
	  if (!enif_get_int(env, argv[6], &n)) return enif_make_int(env,7);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n*c1 * sizeof(float), &c_bin);
	  d = (float *) enif_make_new_binary(env, n*c2 * sizeof(float), &d_bin);
  
  
	  // random-select
	  for(i=0;i<n;i++){
		  r = rand() % r1;
		  for(j=0;j<c1;j++){
			  c[IDX2C(i,j,n)] = a[IDX2C(r,j,r1)];
		  }
		  for(j=0;j<c2;j++){
			  d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
		  }    
	  }
  
	  tuple = enif_make_tuple2(env,c_bin,d_bin);
	  return(tuple);
  }
  
  /*
  random_select for 4D-tensor data
  */
  static ERL_NIF_TERM
  random_select2(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  ERL_NIF_TERM  c_bin,d_bin,tuple;
	  int n1,c1,h1,w1,r2,c2, i, j, k, l, n, r;
	  float *a, *b, *c, *d;
	
	  if (!enif_get_int(env, argv[0], &n1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &c1)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &h1)) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &w1)) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
	  if (!enif_get_int(env, argv[5], &r2)) return enif_make_int(env,6);
	  if (!enif_get_int(env, argv[6], &c2)) return enif_make_int(env,7);
	  if (!enif_inspect_binary(env, argv[7], &b_bin )) return enif_make_int(env,8);
	  if (!enif_get_int(env, argv[8], &n)) return enif_make_int(env,9);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n*c1*h1*w1 * sizeof(float), &c_bin);
	  d = (float *) enif_make_new_binary(env, n*r2*c2 * sizeof(float), &d_bin);
  
	  // random-select
	  for(i=0;i<n;i++){
		  r = rand() % n1;
		  for(j=0;j<c1;j++){
			  for(k=0;k<h1;k++){
				  for(l=0;l<w1;l++){
					  c[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(r,j,k,l,c1,h1,w1)];
				  }
			  }
		  }
		  for(j=0;j<c2;j++){
			  d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
		  }    
	  }
  
	  tuple = enif_make_tuple2(env,c_bin,d_bin);
	  return(tuple);
  }
  
  /*
  random_select for 3D-tensor data
  */
  static ERL_NIF_TERM
  random_select3(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  ERL_NIF_TERM  c_bin,d_bin,tuple;
	  int n1,h1,w1,r2,c2, i, j, k, n, r;
	  float *a, *b, *c, *d;
	
	  if (!enif_get_int(env, argv[0], &n1)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &h1)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &w1)) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &a_bin )) return enif_make_int(env,4);
	  if (!enif_get_int(env, argv[4], &r2)) return enif_make_int(env,5);
	  if (!enif_get_int(env, argv[5], &c2)) return enif_make_int(env,6);
	  if (!enif_inspect_binary(env, argv[6], &b_bin )) return enif_make_int(env,7);
	  if (!enif_get_int(env, argv[7], &n)) return enif_make_int(env,8);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
	  c = (float *) enif_make_new_binary(env, n*h1*w1 * sizeof(float), &c_bin);
	  d = (float *) enif_make_new_binary(env, n*r2*c2 * sizeof(float), &d_bin);
  
	  // random-select
	  for(i=0;i<n;i++){
		  r = rand() % n1;
		  for(j=0;j<h1;j++){
			  for(k=0;k<w1;k++){
				  c[IDX3C(i,j,k,h1,w1)] = a[IDX3C(r,j,k,h1,w1)];
			  }
		  }
		  for(j=0;j<c2;j++){
			  d[IDX2C(i,j,n)] = b[IDX2C(r,j,r2)];
		  }    
	  }
  
	  tuple = enif_make_tuple2(env,c_bin,d_bin);
	  return(tuple);
  }
  
  
  
  static ERL_NIF_TERM
  is_near1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  int i, n, sw;
	  float *a, *b;
	
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
  
	  // near check
	  sw = 0;
	  for(i=0;i<n;i++){
		 if(fabsf(a[i]) > fabsf(b[i])*1.15 || fabsf(a[i]) < fabsf(b[i])*0.85){
			  printf("%f %f \r\n", a[i], b[i]);
			  sw = 1;
		  }
	  }
	  if(sw == 0)
		  return enif_make_int(env,1); //true
	  else
		  return enif_make_int(env,0); //false
  }
  
  static ERL_NIF_TERM
  is_equal1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin,b_bin;
	  int i, n;
	  float *a, *b;
	
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
	  b = (float *) b_bin.data;
  
	  // equal check
	  for(i=0;i<n;i++){
		 if(a[i] != b[i]){
			  return enif_make_int(env,0); //false
		  }
	  }
	  
	  return enif_make_int(env,1); //true
  }
  
  
  
  static ERL_NIF_TERM
  analizer1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  int i, n, id;
	  float *a;
	  float max,min,sum;
	
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &id)) return enif_make_int(env,3);
  
	  a = (float *) a_bin.data;
  
	  // near check
	  for(i=0;i<n;i++){
		  if(isnan(a[i])){
			  return enif_make_int(env,9999);
		  }
		  if(isinf(a[i])){
			  return enif_make_int(env,9998);
		  }
	  }
  
	  //find max min avarage
	  max = -999999999;
	  min = 999999999;
	  sum = 0;
	  for(i=0;i<n;i++){
		  if(a[i] > max)
			  max = a[i];
		  
		  if(a[i] < min)
			  min = a[i];
		  
		  sum = sum+a[i];
  
	  }
	  printf("id max min average\r\n");
	  printf("%d %f %f %f \r\n", id, max, min, sum/(float)n);
  
	  return enif_make_int(env,1);
  }
  
  
  
	
	/*
	1st arg in_n of tensor
	2nd arg in_c of tensor
	3rd arg in_h of tensor
	4th arg in_w of tensor
	5th arg binary of tensor
	
	*/
  static ERL_NIF_TERM
  standardize1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int in_n,in_c,in_h,in_w,n1,i,c1,h1,w1,count;
	  float *a,*b;
	  float sum,average;
	
	  if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &in_h)) return enif_make_int(env,3);
	  if (!enif_get_int(env, argv[3], &in_w)) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &a_bin )) return enif_make_int(env,5);
  
	  n1 = in_n * in_c * in_h * in_w;
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n1 * sizeof(float), &b_bin);
	 
	  
	  for(i=0;i<in_n;i++){
		  sum = 0.0;
		  for(c1=0;c1<in_c;c1++){
			  for(h1=0;h1<in_h;h1++){
				  for(w1=0;w1<in_w;w1++){
					  sum = sum + a[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)];
				  }
			  }
		  }
		  count = in_c * in_h * in_w;
		  average = sum / (float)count;
		  for(c1=0;c1<in_c;c1++){
			  for(h1=0;h1<in_h;h1++){
				  for(w1=0;w1<in_w;w1++){
					  b[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)] = a[IDX4C(i,c1,h1,w1,in_c,in_h,in_w)] - average;
				  }
			  }
		  }
	  }
	  
	  return(b_bin);
  }
  
  
	
  /*
  1st arg in_n of 3D tensor
  2rd arg in_r of 3D tensor
  3th arg in_c of 3D tensor
  4th arg binary of tensor
  5th arg nth in_r of 3D tensor
	
  */
  static ERL_NIF_TERM
  pickup1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int in_n,in_row,in_col,nth,n1,i,j;
	  float *a,*b;
	  
	  if (!enif_get_int(env, argv[0], &in_n)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &in_row)) return enif_make_int(env,2);
	  if (!enif_get_int(env, argv[2], &in_col)) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &a_bin )) return enif_make_int(env,4);
	  if (!enif_get_int(env, argv[4], &nth)) return enif_make_int(env,5);
  
	  n1 = in_n * in_col;
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n1 * sizeof(float), &b_bin);
	   
		
	  for(i=0;i<in_n;i++){
		  for(j=0;j<in_col;j++){
			  b[IDX2C(i,j,in_n)] = a[IDX3C(i,nth,j,in_row,in_col)];
		  }
	  }
		
	  return(b_bin);
  }
	
  
	
  /*
  1st arg size of tensor or matrix
  2rd arg binary of tensor or matrix
	
  */
  static ERL_NIF_TERM
  copy1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin;
	  int n,i;
	  float *a,*b;
	  
	  if (!enif_get_int(env, argv[0], &n)) return enif_make_int(env,1);
	  if (!enif_inspect_binary(env, argv[1], &a_bin )) return enif_make_int(env,2);
  
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
	   
		
	  for(i=0;i<n;i++){
		  b[i] = a[i];
	  }
		
	  return(b_bin);
  }
  
  
  static ERL_NIF_TERM
  slice1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  a_bin;
	  ERL_NIF_TERM  b_bin,c_bin,d_bin,e_bin,tuple;
	  int in_r,in_c,in_c1,i,j,n,bias;
	  float *a,*b,*c,*d,*e;
	  
	  if (!enif_get_int(env, argv[0], &in_r)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &a_bin )) return enif_make_int(env,3);
  
	  in_c1 = in_c / 4;
	  n = in_r * (in_c / 4);
	  a = (float *) a_bin.data;
	  b = (float *) enif_make_new_binary(env, n * sizeof(float), &b_bin);
	  c = (float *) enif_make_new_binary(env, n * sizeof(float), &c_bin);
	  d = (float *) enif_make_new_binary(env, n * sizeof(float), &d_bin);
	  e = (float *) enif_make_new_binary(env, n * sizeof(float), &e_bin);
	   
		
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c1;j++){
			  b[IDX2C(i,j,in_r)] = a[IDX2C(i,j,in_r)]; 
		  }
	  }
	  bias = in_c / 4;
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c1;j++){
			  c[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
		  }
	  }
	  bias = 2 * (in_c / 4);
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c1;j++){
			  d[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
		  }
	  }
	  bias = 3 * (in_c / 4);
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c1;j++){
			  e[IDX2C(i,j,in_r)] = a[IDX2C(i,j+bias,in_r)]; 
		  }
	  }
		
	  tuple = enif_make_tuple4(env,b_bin,c_bin,d_bin,e_bin);
	  return(tuple);
  }
  
  static ERL_NIF_TERM
  unslice1(ErlNifEnv *env, int argc, const ERL_NIF_TERM argv[]) {
	  ErlNifBinary  b_bin,c_bin,d_bin,e_bin;
	  ERL_NIF_TERM  a_bin;
	  int in_r,in_c,i,j,n,bias;
	  float *a,*b,*c,*d,*e;
	  
	  if (!enif_get_int(env, argv[0], &in_r)) return enif_make_int(env,1);
	  if (!enif_get_int(env, argv[1], &in_c)) return enif_make_int(env,2);
	  if (!enif_inspect_binary(env, argv[2], &b_bin )) return enif_make_int(env,3);
	  if (!enif_inspect_binary(env, argv[3], &c_bin )) return enif_make_int(env,4);
	  if (!enif_inspect_binary(env, argv[4], &d_bin )) return enif_make_int(env,5);
	  if (!enif_inspect_binary(env, argv[5], &e_bin )) return enif_make_int(env,6);
  
	  n = in_r * in_c * 4;
	  a = (float *) enif_make_new_binary(env, n * sizeof(float), &a_bin);
	  b = (float *) b_bin.data;
	  c = (float *) c_bin.data;
	  d = (float *) d_bin.data;
	  e = (float *) e_bin.data;
	  
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c;j++){
			  a[IDX2C(i,j,in_r)] = b[IDX2C(i,j,in_r)];
		  }
	  }
	  bias = in_c;
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c;j++){
			  a[IDX2C(i,j+bias,in_r)] = c[IDX2C(i,j,in_r)]; 
		  }
	  }
	  bias = 2 * in_c;
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c;j++){
			  a[IDX2C(i,j+bias,in_r)] = d[IDX2C(i,j,in_r)] ; 
		  }
	  }
	  bias = 3 * in_c;
	  for(i=0;i<in_r;i++){
		  for(j=0;j<in_c;j++){
			  a[IDX2C(i,j+bias,in_r)] = e[IDX2C(i,j,in_r)]; 
		  }
	  }
	  return(a_bin);
  }
  
  
  
  


*/