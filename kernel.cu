#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX3C(c,i,j,in_h,in_w) ((c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define IDX4C(n,c,i,j,in_c,in_h,in_w) ((n)*((in_c)*(in_h)*(in_w)) + (c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define IDX5C(t,n,c,i,j,in_n,in_c,in_h,in_w) ((t)*((in_n)*(in_c)*(in_h)*(in_w)) + (n)*((in_c)*(in_h)*(in_w)) + (c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define SIGMOID(x)  (1 / (1+exp(-1*x)))

#define CHECK(call)                                   \
{                                                     \
    const hipError_t error = call;                   \
    if (error != hipSuccess)                         \
    {                                                 \
        printf("cuda error %d",(int)error);           \
    }                                                 \
}

extern "C" void cuda_add(float *a, float *b, float *c, int n);

__global__ void add1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

void cuda_add(float *a, float *b, float *c, int n);
void cuda_add(float *a, float *b, float *c, int n){
    float *dev_a, *dev_b, *dev_c;

	
    // Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	add1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}


extern "C" void cuda_sub(float *a, float *b, float *c, int n);

__global__ void sub1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] - b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

void cuda_sub(float *a, float *b, float *c, int n);
void cuda_sub(float *a, float *b, float *c, int n){
    float *dev_a, *dev_b, *dev_c;

	
    // Allocate for GPU
	hipMalloc((void**)&dev_a, n * sizeof(float));
	hipMalloc((void**)&dev_b, n * sizeof(float));
	hipMalloc((void**)&dev_c, n * sizeof(float));


    // copy from host a,b to GPU dev_a, dev_b
	hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

	sub1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}


__global__ void pooling_kernel(float *a, float *b, float *c, int st_h, int st_w, int in_c, int in_h, int in_w)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,h1,w1,h2,w2,in_h2,in_w2,start_h1,end_h1,start_w1,end_w1,max_h,max_w;
    float max,fmax_h,fmax_w; 
    n1 = bid;
    c1 = tid;
    in_h2 = in_h / st_h;
    in_w2 = in_w / st_w;
    
    for(w2=0;w2<in_w2;w2++){
        for(h2=0;h2<in_h2;h2++){
            max = -999999999.0;
            start_h1 = st_h*h2;
            end_h1 = st_h*(h2+1);
            start_w1 = st_w*w2;
            end_w1 = st_w*(w2+1);
            for(h1=start_h1;h1<end_h1;h1++){
                for(w1=start_w1;w1<end_w1;w1++){
                    if(a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)] >= max){
                        max = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)];
                        max_h = h1;
                        max_w = w1;
                    }
                }
            }
            b[IDX4C(n1,c1,h2,w2,in_c,in_h2,in_w2)] = max;
            fmax_h = (float)max_h;
            fmax_w = (float)max_w;
            c[IDX4C(n1,c1,h2,w2,in_c,in_h2,in_w2)] = fmax_h * 1000.0 + fmax_w; 
        }
    }
    
}
  
  /*
  1st arg in_n of tensor
  2nd arg in_c of tensor
  3rd arg in_h of tensor
  4th arg in_w of tensor
  5th arg input tensor
  6th arg output tensor (forward result)
  7th arg output tensor (backward result)
  8th arg stride hight
  9th arg stride width

  return list [ts1,ts2]
  ts1 is result data for forward
  ts2 is result data dor backward. this is sparse matrix 
  e.g. 
  |0.1,0.2,0.3,0.4|
  |0.5,0.6,0.7,0.8|
  |0.9,1.0,1.1,1.2|
  |1.3,1.4,1.5,1.6|
  
  ts1
  |0.6,0.8|
  |1.4,1.6|

  ts2
  each element is  row*1000+col
  |1.0*1000+1.0,1.0*1000*3.0|
  |3.0*1000+1.0,3.0*1000+3.0|
  
  */
  extern "C" void pooling1(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c, int st_h, int st_w);
  void pooling1(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c, int st_h, int st_w){
	  int n1, n2;
	  float *dev_a, *dev_b, *dev_c;
	
	  
	  n1 = in_n * in_c * in_h * in_w;
	  n2 = in_n * in_c * (in_h / st_h) * (in_w / st_w);
	  
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n2 * sizeof(float)));
	
	  // copy from host a to GPU dev_a
	  CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
	  
	  dim3 blocks(in_n,1,1);
	  dim3 threads(in_c,1,1);
	  pooling_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, st_h, st_w, in_c, in_h, in_w);
	
	  // copy to host b,c from GPU dev_b,dev_c
	  CHECK(hipMemcpy(b, dev_b, n2 * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(c, dev_c, n2 * sizeof(float), hipMemcpyDeviceToHost));
		
  
	  // return forward data and backward data {b_bin,c_bin} 
	  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);

  }
  
  
  __global__ void unpooling_kernel(float *a, float *b, float *c, int st_h, int st_w, int in_c, int in_h, int in_w)
  {
	  int bid = blockIdx.x;
	  int tid = threadIdx.x;
	  int n1,c1,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1,max_h,max_w,in_h1,in_w1;
	  float loss,elt;
  
	  n1 = bid;
	  c1 = tid;
	  in_h1 = in_h * st_h;
	  in_w1 = in_w * st_w;
	  for(h2=0;h2<in_h;h2++){
		  for(w2=0;w2<in_w;w2++){
			  start_h1 = st_h*h2;
			  end_h1 = st_h*(h2+1);
			  start_w1 = st_w*w2;
			  end_w1 = st_w*(w2+1);
			  elt = a[IDX4C(n1,c1,h2,w2,in_c,in_h,in_w)];
			  loss = b[IDX4C(n1,c1,h2,w2,in_c,in_h,in_w)];
			  max_h = (int) floor(elt / 1000.0);
			  max_w = (int) fmodf(elt,1000.0);
			  for(h1=start_h1;h1<end_h1;h1++){
				  for(w1=start_w1;w1<end_w1;w1++){
					  if(h1 == max_h && w1 == max_w){
						  c[IDX4C(n1,c1,h1,w1,in_c,in_h1,in_w1)] = loss;
					  }
					  else{
						  c[IDX4C(n1,c1,h1,w1,in_c,in_h1,in_w1)] = 0.0;
					  }
				  }
			  }
		  }
	  }
		  
  }
	
  /*
  1st arg in_n of sparse-tensor
  2nd arg in_c of sparse-tensor
  3rd arg in_h of sparse-tensor
  4th arg in_w of sparse-tensor
  5th arg input sparse-tensor
  6th arg input loss-tensor
  7th arg output tensor(backward)
  8th arg stride hight
  9th arg stride width
  
  return gradiate tensor
  e.g.
  ts1 index-tensor
	each element is  row*1000+col
	|1.0*1000+1.0,1.0*1000*3.0|
	|3.0*1000+1.0,3.0*1000+3.0|
  ts2 loss-tensor
	|0.1,0.2|
	|0.3,0.4|
  
  return
	|0.0,0.0,0.0,0.0|
	|0.0,0.1,0.0,0.2|
	|0.0,0.0,0.0,0.0|
	|0.0,3.4,0.0,0.4|
  
  */
  extern "C" void unpooling1(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c,int st_h, int st_w);
  void unpooling1(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c,int st_h, int st_w){
	  int n1, n2;
	  float *dev_a, *dev_b, *dev_c;
	
	  n1 = in_n * in_c * in_h * in_w;
	  n2 = in_n * in_c * (in_h * st_h) * (in_w * st_w);
	  
		
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n2 * sizeof(float)));
  
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

	  
	  dim3 blocks(in_n,1,1);
	  dim3 threads(in_c,1,1);
	  unpooling_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, st_h, st_w, in_c, in_h, in_w);
	
	  // copy to host d from GPU dev_d
	  CHECK(hipMemcpy(c, dev_c, n2 * sizeof(float), hipMemcpyDeviceToHost));
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
  
  }
  
  	
  
  __global__ void convolute1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w,
	int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int oh, int ow)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1;
   float sum,elt1,elt2;
	 
   n1 = bid;
   c2 = tid;
   for(w2=0;w2<ow;w2++){
	   for(h2=0;h2<oh;h2++){
		   sum = 0.0;
		   start_h1 = st_h*h2-pad;
		   end_h1 = start_h1 + filt_h;
		   start_w1 = st_w*w2-pad;
		   end_w1 = start_w1 + filt_w;
		   for(c1=0;c1<in_c;c1++){
			   for(h1=start_h1;h1<end_h1;h1++){
				   for(w1=start_w1;w1<end_w1;w1++){
					   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
						   elt1 = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)];
						   elt2 = b[IDX4C(c2,c1,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)];
						   sum = sum + elt1*elt2;
					   }
				   }
			   }
		   }
		   c[IDX4C(n1,c2,h2,w2,filt_n,oh,ow)] = sum;   
	   }
   }
	   
}
 
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input tensor
10th arg filter tensor
11th arg filter output-tensor
12th arg stride hight
13th arg stride width
14th arg padding   
*/
extern "C" void convolute1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, float *a, float *b, float *c,
	int st_h, int st_w, int pad);

void convolute1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, float *a, float *b, float *c,
                int st_h, int st_w, int pad){
   int n1, n2, n3, oh, ow;
   float *dev_a, *dev_b, *dev_c;
 
   
   n1 = in_n * in_c * in_h * in_w;
   n2 = filt_n * filt_c * filt_h * filt_w;
   oh = (in_h+2*pad-filt_h)/st_h + 1;
   ow = (in_w+2*pad-filt_w)/st_w + 1;
   n3 = in_n * filt_n * oh * ow;  // n of filter generate n channel
   
   // Allocate for GPU
   CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

 
   // copy from host a,b to GPU dev_a, dev_b
   CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));


   dim3 blocks(in_n,1,1);
   dim3 threads(filt_n,1,1);
   convolute1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad, in_c, in_h, in_w, oh, ow);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

   // free 
   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   
}


  
__global__ void deconvolute1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w,
	int st_h, int st_w, int pad1, int pad, int in_c, int in_h, int in_w, int oh, int ow, int oh1, int ow1)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1;
   float sum,elt1,elt2;
   
   n1 = bid;
   c2 = tid;
	   
   //full convolute. stride=1 always
   for(w2=0;w2<ow;w2++){
	   for(h2=0;h2<oh;h2++){
		   start_h1 = h2-pad1;  
		   end_h1 = start_h1 + filt_h;
		   start_w1 = w2-pad1;
		   end_w1 = start_w1 + filt_w;
		   sum = 0.0;
		   for(h1=start_h1;h1<end_h1;h1++){
			   for(w1=start_w1;w1<end_w1;w1++){
				   for(c1=0;c1<filt_n;c1++){        
					   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
						   elt1 = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)]; //loss tensor
						   elt2 = b[IDX4C(c1,c2,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)]; //filter tensor
						   sum = sum + elt1*elt2;
					   } 
				   }   
			   }
		   }
		   if(h2-pad >=0 && h2-pad < oh1 && w2-pad >= 0 && w2-pad < ow1){
			   c[IDX4C(n1,c2,h2-pad,w2-pad,filt_c,oh1,ow1)] = sum;
		   }             
	   }
   }
   
}
 
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input loss tensor
10th arg filter tensor
11th arg output tensor
12th arg stride hight
13th arg stride width
14th arg padding   

memo
ex padding = 1
loss 4*4
filter 2*2
input 3*3  padding=1
(3-2+2*1)/1 + 1 = 4  
decovolute compute 5*5(3*3 padding=1) and save result range 3*3


*/
void deconvolute1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w,
                   float *a, float *b, float *c, int st_h, int st_w, int pad){
   int pad1, n1, n2, n3, oh, ow, oh1, ow1, i,j,k,l;
   float *b1;
   float *dev_a, *dev_b, *dev_c;
 
   
   n1 = in_n * in_c * in_h * in_w;
   n2 = filt_n * filt_c * filt_h * filt_w;
   pad1 = filt_h - 1;
   // pad1 = filt_h -1,  pad is original padding size
   oh = (in_h+2*pad1-filt_h)/st_h + 1;
   ow = (in_w+2*pad1-filt_w)/st_w + 1;
   oh1 = (in_h+2*(pad1-pad)-filt_h)/st_h + 1;
   ow1 = (in_w+2*(pad1-pad)-filt_w)/st_w + 1;
   n3 = in_n * filt_c * oh1 * ow1;  // channel of filter generate same channel input tensor
   b1 = (float *)malloc (n2 * sizeof (float));
 
	 
   //rotate 180 degree
   for(i=0;i<filt_n;i++){  
	   for(j=0;j<filt_c;j++){
		   for(k=0;k<filt_h;k++){
			   for(l=0;l<filt_w;l++){
				   b1[IDX4C(i,j,filt_h-k-1,filt_w-l-1,filt_c,filt_h,filt_w)] = b[IDX4C(i,j,k,l,filt_c,filt_h,filt_w)];
			   }
		   }
	   }
   }

   
   // Allocate for GPU
   CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

 
   // copy from host a,b1,c to GPU dev_a, dev_b, dev_c
   CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b1, n2 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

   dim3 blocks(in_n,1,1);
   dim3 threads(filt_c,1,1);
   deconvolute1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad1, pad, in_c, in_h, in_w, oh, ow, oh1, ow1);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

   // free 
   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   free(b1);
   
}




__global__ void deconvolute2_kernel(float *a1, float *a, float *b, float *c, int filt_n, int filt_c,int filt_h, int filt_w,
	int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int loss_h, int loss_w)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,oh,ow,start_h1,end_h1,start_w1,end_w1;
   int j,k,l,k1,l1;
   float sum,elt1,elt2;
   
   n1 = bid;
   c2 = tid;
   // caution! stride=1 
   oh = (in_h+2*pad-filt_h) + 1;
   ow = (in_w+2*pad-filt_w) + 1;
   
   //dilate loss tensor.
   for(j=0;j<filt_n;j++){
	   for(k=0;k<loss_h;k++){
		   for(l=0;l<loss_w;l++){
			   elt1 = a[IDX4C(n1,j,k,l,in_c,loss_h,loss_w)];
			   k1 = st_h*k;
			   l1 = st_w*l;
			   a1[IDX4C(n1,j,k1,l1,in_c,in_h,in_w)] = elt1;
		   }
	   }
   }
   //full convulute. stride=1
   for(c2=0;c2<filt_c;c2++){
	   for(w2=0;w2<ow;w2++){
		   for(h2=0;h2<oh;h2++){
			   start_h1 = h2-pad;
			   end_h1 = start_h1 + filt_h;
			   start_w1 = w2-pad;
			   end_w1 = start_w1 + filt_w;
			   sum = 0.0;
			   for(h1=start_h1;h1<end_h1;h1++){
				   for(w1=start_w1;w1<end_w1;w1++){
					   for(c1=0;c1<filt_n;c1++){        
						   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
							   elt1 = a1[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)]; //loss tensor
							   elt2 = b[IDX4C(c1,c2,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)]; //filter tensor
							   sum = sum + elt1*elt2;
						   }
					   }
				   }   
			   }
			   c[IDX4C(n1,c2,h2,w2,filt_c,oh,ow)] = sum;              
		   }
	   }
   }
   
}




/*
dilate loss tensor 
e.g.

|1.0,2.0|
|3.0,4.0|

dilated stride=2
|1.0,0.0,2.0|
|0.0,0.0,0.0|
|3.0,0.0,4.0|


*/


/*
1st arg in_n of input loss tensor
2nd arg in_c of input loss tensor
3rd arg in_h of input loss  tensor
4th arg in_w of input loss tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input loss tensor
10th arg filter tensor
11th arg output tensor
12th arg stride hight
13th arg stride width
14th arg padding   
*/

void deconvolute2(int in_n, int in_c, int loss_h, int loss_w, int filt_n, int filt_c, int filt_h,int filt_w,
             float *a, float *b, float *c,int st_h, int st_w,int pad){
   int pad1, n1, n2, n3, oh, ow, i,j,k,l,in_h, in_w;
   float *a1, *b1;
   float *dev_a, *dev_a1, *dev_b, *dev_c;

 
	   
   // size for dilate
   in_h = loss_h + (loss_h - 1)*(st_h - 1);
   in_w = loss_w + (loss_w - 1)*(st_w - 1);

   n1 = in_n * in_c * in_h * in_w;  //loss tensor size 
   n2 = filt_n * filt_c * filt_h * filt_w;  //filter tensor size
   pad1 = (filt_h - 1) + pad;    //padding size with dilate
   oh = (in_h+2*pad1-filt_h) + 1; //output deconvolute tensor size. caution stride=1.
   ow = (in_w+2*pad1-filt_w) + 1; // 
   n3 = in_n * filt_c * oh * ow;   // 
   a1 = (float *)malloc (n1 * sizeof (float));
   b1 = (float *)malloc (n2 * sizeof (float));

   //rotate 180 degree
   for(i=0;i<filt_n;i++){  
	   for(j=0;j<filt_c;j++){
		   for(k=0;k<filt_h;k++){
			   for(l=0;l<filt_w;l++){
				   b1[IDX4C(i,j,filt_h-k-1,filt_w-l-1,filt_c,filt_h,filt_w)] = b[IDX4C(i,j,k,l,filt_c,filt_h,filt_w)];
			   }
		   }
	   }
   }


   // dilate 
   for(i=0;i<n1;i++){
	   a1[i] = 0.0;
   }

   CHECK(hipMalloc((void**)&dev_a1, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_a, in_n*1*loss_h*loss_w * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

   CHECK(hipMemcpy(dev_a1, a1, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_a, a, in_n*1*loss_h*loss_w  * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b1, n2 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

   dim3 blocks(in_n,1,1);
   dim3 threads(filt_c,1,1);
   deconvolute2_kernel <<<blocks, filt_c>> >(dev_a1, dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad1, in_c, in_h, in_w, loss_h, loss_w);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));
   
   // free 
   hipFree(dev_a);
   hipFree(dev_a1);
   hipFree(dev_b);
   hipFree(dev_c);
   free(a1);
   free(b1);
 
}

