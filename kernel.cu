#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define CHECK(call)                                   \
{                                                     \
    const hipError_t error = call;                   \
    if (error != hipSuccess)                         \
    {                                                 \
        return(10000+(int)error);   \
    }                                                 \
}

extern "C" void cuda_add(float *a, float *b, float *c, int n);

__global__ void add1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

void cuda_add(float *a, float *b, float *c, int n);
void cuda_add(float *a, float *b, float *c, int n){
    float *dev_a, *dev_b, *dev_c;

    // Allocate for GPU
	hipMalloc((void**)&dev_a, n * sizeof(float));
	hipMalloc((void**)&dev_b, n * sizeof(float));
	hipMalloc((void**)&dev_c, n * sizeof(float));


    // copy from host a,b to GPU dev_a, dev_b
	hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

	add1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}
