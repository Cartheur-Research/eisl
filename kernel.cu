#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX3C(c,i,j,in_h,in_w) ((c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define IDX4C(n,c,i,j,in_c,in_h,in_w) ((n)*((in_c)*(in_h)*(in_w)) + (c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define IDX5C(t,n,c,i,j,in_n,in_c,in_h,in_w) ((t)*((in_n)*(in_c)*(in_h)*(in_w)) + (n)*((in_c)*(in_h)*(in_w)) + (c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define SIGMOID(x)  (1 / (1+exp(-1*x)))

#define CHECK(call)                                   \
{                                                     \
    const hipError_t error = call;                   \
    if (error != hipSuccess)                         \
    {                                                 \
        printf("cuda error %d",(int)error);           \
    }                                                 \
}

extern "C" void cuda_add(float *a, float *b, float *c, int n);

__global__ void add1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

void cuda_add(float *a, float *b, float *c, int n);
void cuda_add(float *a, float *b, float *c, int n){
    float *dev_a, *dev_b, *dev_c;

	
    // Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	add1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}


extern "C" void cuda_sub(float *a, float *b, float *c, int n);

__global__ void sub1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] - b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

void cuda_sub(float *a, float *b, float *c, int n);
void cuda_sub(float *a, float *b, float *c, int n){
    float *dev_a, *dev_b, *dev_c;

	
    // Allocate for GPU
	hipMalloc((void**)&dev_a, n * sizeof(float));
	hipMalloc((void**)&dev_b, n * sizeof(float));
	hipMalloc((void**)&dev_c, n * sizeof(float));


    // copy from host a,b to GPU dev_a, dev_b
	hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

	sub1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}


__global__ void pooling_kernel(float *a, float *b, float *c, int st_h, int st_w, int in_c, int in_h, int in_w)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,h1,w1,h2,w2,in_h2,in_w2,start_h1,end_h1,start_w1,end_w1,max_h,max_w;
    float max,fmax_h,fmax_w; 
    n1 = bid;
    c1 = tid;
    in_h2 = in_h / st_h;
    in_w2 = in_w / st_w;
    
    for(w2=0;w2<in_w2;w2++){
        for(h2=0;h2<in_h2;h2++){
            max = -999999999.0;
            start_h1 = st_h*h2;
            end_h1 = st_h*(h2+1);
            start_w1 = st_w*w2;
            end_w1 = st_w*(w2+1);
            for(h1=start_h1;h1<end_h1;h1++){
                for(w1=start_w1;w1<end_w1;w1++){
                    if(a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)] >= max){
                        max = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)];
                        max_h = h1;
                        max_w = w1;
                    }
                }
            }
            b[IDX4C(n1,c1,h2,w2,in_c,in_h2,in_w2)] = max;
            fmax_h = (float)max_h;
            fmax_w = (float)max_w;
            c[IDX4C(n1,c1,h2,w2,in_c,in_h2,in_w2)] = fmax_h * 1000.0 + fmax_w; 
        }
    }
    
}
  
  /*
  1st arg in_n of tensor
  2nd arg in_c of tensor
  3rd arg in_h of tensor
  4th arg in_w of tensor
  5th arg input tensor
  6th arg output tensor (forward result)
  7th arg output tensor (backward result)
  8th arg stride hight
  9th arg stride width

  return list [ts1,ts2]
  ts1 is result data for forward
  ts2 is result data dor backward. this is sparse matrix 
  e.g. 
  |0.1,0.2,0.3,0.4|
  |0.5,0.6,0.7,0.8|
  |0.9,1.0,1.1,1.2|
  |1.3,1.4,1.5,1.6|
  
  ts1
  |0.6,0.8|
  |1.4,1.6|

  ts2
  each element is  row*1000+col
  |1.0*1000+1.0,1.0*1000*3.0|
  |3.0*1000+1.0,3.0*1000+3.0|
  
  */
  extern "C" void pooling1(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c, int st_h, int st_w);
  void pooling1(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c, int st_h, int st_w){
	  int n1, n2;
	  float *dev_a, *dev_b, *dev_c;
	
	  
	  n1 = in_n * in_c * in_h * in_w;
	  n2 = in_n * in_c * (in_h / st_h) * (in_w / st_w);
	  
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n2 * sizeof(float)));
	
	  // copy from host a to GPU dev_a
	  CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
	  
	  dim3 blocks(in_n,1,1);
	  dim3 threads(in_c,1,1);
	  pooling_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, st_h, st_w, in_c, in_h, in_w);
	
	  // copy to host b,c from GPU dev_b,dev_c
	  CHECK(hipMemcpy(b, dev_b, n2 * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(c, dev_c, n2 * sizeof(float), hipMemcpyDeviceToHost));
		
  
	  // return forward data and backward data {b_bin,c_bin} 
	  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);

  }
  
  
  __global__ void unpooling_kernel(float *a, float *b, float *c, int st_h, int st_w, int in_c, int in_h, int in_w)
  {
	  int bid = blockIdx.x;
	  int tid = threadIdx.x;
	  int n1,c1,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1,max_h,max_w,in_h1,in_w1;
	  float loss,elt;
  
	  n1 = bid;
	  c1 = tid;
	  in_h1 = in_h * st_h;
	  in_w1 = in_w * st_w;
	  for(h2=0;h2<in_h;h2++){
		  for(w2=0;w2<in_w;w2++){
			  start_h1 = st_h*h2;
			  end_h1 = st_h*(h2+1);
			  start_w1 = st_w*w2;
			  end_w1 = st_w*(w2+1);
			  elt = a[IDX4C(n1,c1,h2,w2,in_c,in_h,in_w)];
			  loss = b[IDX4C(n1,c1,h2,w2,in_c,in_h,in_w)];
			  max_h = (int) floor(elt / 1000.0);
			  max_w = (int) fmodf(elt,1000.0);
			  for(h1=start_h1;h1<end_h1;h1++){
				  for(w1=start_w1;w1<end_w1;w1++){
					  if(h1 == max_h && w1 == max_w){
						  c[IDX4C(n1,c1,h1,w1,in_c,in_h1,in_w1)] = loss;
					  }
					  else{
						  c[IDX4C(n1,c1,h1,w1,in_c,in_h1,in_w1)] = 0.0;
					  }
				  }
			  }
		  }
	  }
		  
  }
	
  /*
  1st arg in_n of sparse-tensor
  2nd arg in_c of sparse-tensor
  3rd arg in_h of sparse-tensor
  4th arg in_w of sparse-tensor
  5th arg input sparse-tensor
  6th arg input loss-tensor
  7th arg output tensor(backward)
  8th arg stride hight
  9th arg stride width
  
  return gradiate tensor
  e.g.
  ts1 index-tensor
	each element is  row*1000+col
	|1.0*1000+1.0,1.0*1000*3.0|
	|3.0*1000+1.0,3.0*1000+3.0|
  ts2 loss-tensor
	|0.1,0.2|
	|0.3,0.4|
  
  return
	|0.0,0.0,0.0,0.0|
	|0.0,0.1,0.0,0.2|
	|0.0,0.0,0.0,0.0|
	|0.0,3.4,0.0,0.4|
  
  */
  extern "C" void unpooling1(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c,int st_h, int st_w);
  void unpooling1(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c,int st_h, int st_w){
	  int n1, n2;
	  float *dev_a, *dev_b, *dev_c;
	
	  n1 = in_n * in_c * in_h * in_w;
	  n2 = in_n * in_c * (in_h * st_h) * (in_w * st_w);
	  
		
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n2 * sizeof(float)));
  
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

	  
	  dim3 blocks(in_n,1,1);
	  dim3 threads(in_c,1,1);
	  unpooling_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, st_h, st_w, in_c, in_h, in_w);
	
	  // copy to host d from GPU dev_d
	  CHECK(hipMemcpy(c, dev_c, n2 * sizeof(float), hipMemcpyDeviceToHost));
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
  
  }
  
  	
  
  __global__ void convolute1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w,
	int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int oh, int ow)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1;
   float sum,elt1,elt2;
	 
   n1 = bid;
   c2 = tid;
   for(w2=0;w2<ow;w2++){
	   for(h2=0;h2<oh;h2++){
		   sum = 0.0;
		   start_h1 = st_h*h2-pad;
		   end_h1 = start_h1 + filt_h;
		   start_w1 = st_w*w2-pad;
		   end_w1 = start_w1 + filt_w;
		   for(c1=0;c1<in_c;c1++){
			   for(h1=start_h1;h1<end_h1;h1++){
				   for(w1=start_w1;w1<end_w1;w1++){
					   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
						   elt1 = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)];
						   elt2 = b[IDX4C(c2,c1,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)];
						   sum = sum + elt1*elt2;
					   }
				   }
			   }
		   }
		   c[IDX4C(n1,c2,h2,w2,filt_n,oh,ow)] = sum;   
	   }
   }
	   
}
 
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input tensor
10th arg filter tensor
11th arg filter output-tensor
12th arg stride hight
13th arg stride width
14th arg padding   
*/
extern "C" void convolute1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, float *a, float *b, float *c,
	int st_h, int st_w, int pad);

void convolute1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, float *a, float *b, float *c,
                int st_h, int st_w, int pad){
   int n1, n2, n3, oh, ow;
   float *dev_a, *dev_b, *dev_c;
 
   
   n1 = in_n * in_c * in_h * in_w;
   n2 = filt_n * filt_c * filt_h * filt_w;
   oh = (in_h+2*pad-filt_h)/st_h + 1;
   ow = (in_w+2*pad-filt_w)/st_w + 1;
   n3 = in_n * filt_n * oh * ow;  // n of filter generate n channel
   
   // Allocate for GPU
   CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

 
   // copy from host a,b to GPU dev_a, dev_b
   CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));


   dim3 blocks(in_n,1,1);
   dim3 threads(filt_n,1,1);
   convolute1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad, in_c, in_h, in_w, oh, ow);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

   // free 
   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   
}


  
__global__ void deconvolute1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w,
	int st_h, int st_w, int pad1, int pad, int in_c, int in_h, int in_w, int oh, int ow, int oh1, int ow1)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1;
   float sum,elt1,elt2;
   
   n1 = bid;
   c2 = tid;
	   
   //full convolute. stride=1 always
   for(w2=0;w2<ow;w2++){
	   for(h2=0;h2<oh;h2++){
		   start_h1 = h2-pad1;  
		   end_h1 = start_h1 + filt_h;
		   start_w1 = w2-pad1;
		   end_w1 = start_w1 + filt_w;
		   sum = 0.0;
		   for(h1=start_h1;h1<end_h1;h1++){
			   for(w1=start_w1;w1<end_w1;w1++){
				   for(c1=0;c1<filt_n;c1++){        
					   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
						   elt1 = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)]; //loss tensor
						   elt2 = b[IDX4C(c1,c2,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)]; //filter tensor
						   sum = sum + elt1*elt2;
					   } 
				   }   
			   }
		   }
		   if(h2-pad >=0 && h2-pad < oh1 && w2-pad >= 0 && w2-pad < ow1){
			   c[IDX4C(n1,c2,h2-pad,w2-pad,filt_c,oh1,ow1)] = sum;
		   }             
	   }
   }
   
}
 
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input loss tensor
10th arg filter tensor
11th arg output tensor
12th arg stride hight
13th arg stride width
14th arg padding   

memo
ex padding = 1
loss 4*4
filter 2*2
input 3*3  padding=1
(3-2+2*1)/1 + 1 = 4  
decovolute compute 5*5(3*3 padding=1) and save result range 3*3


*/
void deconvolute1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w,
                   float *a, float *b, float *c, int st_h, int st_w, int pad){
   int pad1, n1, n2, n3, oh, ow, oh1, ow1, i,j,k,l;
   float *b1;
   float *dev_a, *dev_b, *dev_c;
 
   
   n1 = in_n * in_c * in_h * in_w;
   n2 = filt_n * filt_c * filt_h * filt_w;
   pad1 = filt_h - 1;
   // pad1 = filt_h -1,  pad is original padding size
   oh = (in_h+2*pad1-filt_h)/st_h + 1;
   ow = (in_w+2*pad1-filt_w)/st_w + 1;
   oh1 = (in_h+2*(pad1-pad)-filt_h)/st_h + 1;
   ow1 = (in_w+2*(pad1-pad)-filt_w)/st_w + 1;
   n3 = in_n * filt_c * oh1 * ow1;  // channel of filter generate same channel input tensor
   b1 = (float *)malloc (n2 * sizeof (float));
 
	 
   //rotate 180 degree
   for(i=0;i<filt_n;i++){  
	   for(j=0;j<filt_c;j++){
		   for(k=0;k<filt_h;k++){
			   for(l=0;l<filt_w;l++){
				   b1[IDX4C(i,j,filt_h-k-1,filt_w-l-1,filt_c,filt_h,filt_w)] = b[IDX4C(i,j,k,l,filt_c,filt_h,filt_w)];
			   }
		   }
	   }
   }

   
   // Allocate for GPU
   CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

 
   // copy from host a,b1,c to GPU dev_a, dev_b, dev_c
   CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b1, n2 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

   dim3 blocks(in_n,1,1);
   dim3 threads(filt_c,1,1);
   deconvolute1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad1, pad, in_c, in_h, in_w, oh, ow, oh1, ow1);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

   // free 
   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   free(b1);
   
}




__global__ void deconvolute2_kernel(float *a1, float *a, float *b, float *c, int filt_n, int filt_c,int filt_h, int filt_w,
	int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int loss_h, int loss_w)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,oh,ow,start_h1,end_h1,start_w1,end_w1;
   int j,k,l,k1,l1;
   float sum,elt1,elt2;
   
   n1 = bid;
   c2 = tid;
   // caution! stride=1 
   oh = (in_h+2*pad-filt_h) + 1;
   ow = (in_w+2*pad-filt_w) + 1;
   
   //dilate loss tensor.
   for(j=0;j<filt_n;j++){
	   for(k=0;k<loss_h;k++){
		   for(l=0;l<loss_w;l++){
			   elt1 = a[IDX4C(n1,j,k,l,in_c,loss_h,loss_w)];
			   k1 = st_h*k;
			   l1 = st_w*l;
			   a1[IDX4C(n1,j,k1,l1,in_c,in_h,in_w)] = elt1;
		   }
	   }
   }
   //full convulute. stride=1
   for(c2=0;c2<filt_c;c2++){
	   for(w2=0;w2<ow;w2++){
		   for(h2=0;h2<oh;h2++){
			   start_h1 = h2-pad;
			   end_h1 = start_h1 + filt_h;
			   start_w1 = w2-pad;
			   end_w1 = start_w1 + filt_w;
			   sum = 0.0;
			   for(h1=start_h1;h1<end_h1;h1++){
				   for(w1=start_w1;w1<end_w1;w1++){
					   for(c1=0;c1<filt_n;c1++){        
						   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
							   elt1 = a1[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)]; //loss tensor
							   elt2 = b[IDX4C(c1,c2,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)]; //filter tensor
							   sum = sum + elt1*elt2;
						   }
					   }
				   }   
			   }
			   c[IDX4C(n1,c2,h2,w2,filt_c,oh,ow)] = sum;              
		   }
	   }
   }
   
}




/*
dilate loss tensor 
e.g.

|1.0,2.0|
|3.0,4.0|

dilated stride=2
|1.0,0.0,2.0|
|0.0,0.0,0.0|
|3.0,0.0,4.0|


*/


/*
1st arg in_n of input loss tensor
2nd arg in_c of input loss tensor
3rd arg in_h of input loss  tensor
4th arg in_w of input loss tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input loss tensor
10th arg filter tensor
11th arg output tensor
12th arg stride hight
13th arg stride width
14th arg padding   
*/

void deconvolute2(int in_n, int in_c, int loss_h, int loss_w, int filt_n, int filt_c, int filt_h,int filt_w,
             float *a, float *b, float *c,int st_h, int st_w,int pad){
   int pad1, n1, n2, n3, oh, ow, i,j,k,l,in_h, in_w;
   float *a1, *b1;
   float *dev_a, *dev_a1, *dev_b, *dev_c;

 
	   
   // size for dilate
   in_h = loss_h + (loss_h - 1)*(st_h - 1);
   in_w = loss_w + (loss_w - 1)*(st_w - 1);

   n1 = in_n * in_c * in_h * in_w;  //loss tensor size 
   n2 = filt_n * filt_c * filt_h * filt_w;  //filter tensor size
   pad1 = (filt_h - 1) + pad;    //padding size with dilate
   oh = (in_h+2*pad1-filt_h) + 1; //output deconvolute tensor size. caution stride=1.
   ow = (in_w+2*pad1-filt_w) + 1; // 
   n3 = in_n * filt_c * oh * ow;   // 
   a1 = (float *)malloc (n1 * sizeof (float));
   b1 = (float *)malloc (n2 * sizeof (float));

   //rotate 180 degree
   for(i=0;i<filt_n;i++){  
	   for(j=0;j<filt_c;j++){
		   for(k=0;k<filt_h;k++){
			   for(l=0;l<filt_w;l++){
				   b1[IDX4C(i,j,filt_h-k-1,filt_w-l-1,filt_c,filt_h,filt_w)] = b[IDX4C(i,j,k,l,filt_c,filt_h,filt_w)];
			   }
		   }
	   }
   }


   // dilate 
   for(i=0;i<n1;i++){
	   a1[i] = 0.0;
   }

   CHECK(hipMalloc((void**)&dev_a1, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_a, in_n*1*loss_h*loss_w * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

   CHECK(hipMemcpy(dev_a1, a1, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_a, a, in_n*1*loss_h*loss_w  * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b1, n2 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

   dim3 blocks(in_n,1,1);
   dim3 threads(filt_c,1,1);
   deconvolute2_kernel <<<blocks, filt_c>> >(dev_a1, dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad1, in_c, in_h, in_w, loss_h, loss_w);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));
   
   // free 
   hipFree(dev_a);
   hipFree(dev_a1);
   hipFree(dev_b);
   hipFree(dev_c);
   free(a1);
   free(b1);
 
}

  
__global__ void gradfilter1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w, int loss_c, int loss_h, int loss_w, int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int n)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,h3,w3;
    float sum,elt1,elt2;

    n1 = bid;
    c2 = tid;
    
    for(c1=0;c1<filt_c;c1++){
    //h1,w1 is index of filter
        for(h1=0;h1<filt_h;h1++){
            for(w1=0;w1<filt_w;w1++){
                //h2,w2 is index of loss tensor
                sum = 0.0;
                for(h2=0;h2<loss_h;h2++){
                    for(w2=0;w2<loss_w;w2++){
                        //h3,w3 is index of input tensor
                        h3 = h1 - pad + h2;
                        w3 = w1 - pad + w2;
                        if(h3>=0 && h3<in_h && w3>=0 && w3<in_w){
                            elt1 = a[IDX4C(n1,c1,h3,w3,in_c,in_h,in_w)];    //input tensor
                            elt2 = b[IDX4C(n1,c2,h2,w2,loss_c,loss_h,loss_w)]; //loss tensor
                            sum = sum + elt1*elt2;
                        }
                    }
                }
                //set filter tensor
                c[IDX5C(n1,c2,c1,h1,w1,filt_n,filt_c,filt_h,filt_w)] =  sum;
            }
        }
    } 
               
}



  
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg loss_c of loss tensor
10th arg loss_h of loss tensor
11th arg loss_w of loss tensor
12th arg filter tensor
13th arg loss tensor
14th arg output tensor
15th arg stride hight
16th arg stride width
17th arg padding   
*/

void gradfilter1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w,
                 int loss_c, int loss_h, int loss_w, float *a, float *b, float *d, int st_h, int st_w, int pad){
    int n1,n2,n3,n4,i,j,k,l,m;
    float *c;
    float *dev_a, *dev_b, *dev_c;
    float elt;
  
    
    n1 = in_n * in_c * in_h * in_w;
    n2 = in_n * loss_c * loss_h * loss_w;
    n3 = in_n * filt_n * filt_c * filt_h * filt_w;
    n4 = filt_n * filt_c * filt_h * filt_w;
    c = (float *)malloc (n3 * sizeof (float));

    //initialize c
    for(i=0;i<n3;i++){
        c[i] = 0.0;
    }
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

    
    // copy from host a,b,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_n,1,1);
    gradfilter1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, loss_c, loss_h, loss_w, st_h, st_w, pad, in_c, in_h, in_w, in_n);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

    //average
    // clear d
    for(i=0;i<n4;i++){
        d[i] = 0.0;
    }
    // copy from c to d and compute sum
    for(i=0;i<in_n;i++){
        for(j=0;j<filt_n;j++){
            for(k=0;k<filt_c;k++){
                for(l=0;l<filt_h;l++){
                    for(m=0;m<filt_w;m++){
                        elt = c[IDX5C(i,j,k,l,m,filt_n,filt_c,filt_h,filt_w)];
                        d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] = d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] + elt;
                    }
                }
            }
        }
    }
    // average
    for(i=0;i<n4;i++){
        d[i] = d[i] / (float)in_n;
    }
    
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(c);
}

  
__global__ void gradfilter2_kernel(float *a, float *b1, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w, int loss_c, int loss_h, int loss_w, int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int n)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,h3,w3,loss_h1,loss_w1,j,k,l,k1,l1;
    float sum,elt1,elt2;
    
    n1 = bid;
    c2 = tid;
    //dilated loss tensor size
    loss_h1 = loss_h+(loss_h-1)*(st_h-1);
    loss_w1 = loss_w+(loss_w-1)*(st_w-1);
    //dilate loss tensor.
    for(j=0;j<loss_c;j++){
        for(k=0;k<loss_h;k++){
            for(l=0;l<loss_w;l++){
                elt1 = b[IDX4C(n1,j,k,l,loss_c,loss_h,loss_w)];
                k1 = st_h*k;
                l1 = st_w*l;
                b1[IDX4C(n1,j,k1,l1,loss_c,loss_h1,loss_w1)] = elt1;
            }
        }
    }
    //convolute input tensor with dilated loss tensor. cuation stride is always 1. 
    for(c1=0;c1<filt_c;c1++){
    //h1,w1 is index of filter
        for(h1=0;h1<filt_h;h1++){
            for(w1=0;w1<filt_w;w1++){
                //h2,w2 is index of loss tensor
                sum = 0.0;
                for(h2=0;h2<loss_h1;h2++){
                    for(w2=0;w2<loss_w1;w2++){
                        //h3,w3 is index of input tensor
                        h3 = h1 - pad + h2;
                        w3 = w1 - pad + w2;
                        if(h3>=0 && h3<in_h && w3>=0 && w3<in_w){
                            elt1 = a[IDX4C(n1,c1,h3,w3,in_c,in_h,in_w)];    //input tensor
                            elt2 = b1[IDX4C(n1,c2,h2,w2,loss_c,loss_h1,loss_w1)]; //loss tensor
                            sum = sum + elt1*elt2;
                        }
                    }
                }
                //set filter tensor
                c[IDX5C(n1,c2,c1,h1,w1,filt_n,filt_c,filt_h,filt_w)] = + sum;
            }
        }
    } 
        
}

/*
dilate loss tensor 
e.g.

|1.0,2.0|
|3.0,4.0|

dilated stride=2
|1.0,0.0,2.0|
|0.0,0.0,0.0|
|3.0,0.0,4.0|


*/

/*
gradfilter2 is for stride >= 2. This one requires dilate
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg loss_c of loss tensor
10th arg loss_h of loss tensor
11th arg loss_w of loss tensor
12th arg filter tensor
13th arg loss tensor
14th arg output tensor
15th arg stride hight
16th arg stride width
17th arg padding  

*/
void gradfilter2(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, 
                 int loss_c, int loss_h, int loss_w, float *a, float *b, float *d, int st_h, int st_w, int pad){
    int n1,n2,n3,n4,n5,i,j,k,l,m;
    float *b1,*c;
    float *dev_a, *dev_b, *dev_b1, *dev_c;
    float elt;
  
    n1 = in_n * in_c * in_h * in_w;
    n2 = in_n * loss_c * loss_h * loss_w;
    n3 = in_n * filt_n * filt_c * filt_h * filt_w;
    n4 = filt_n * filt_c * filt_h * filt_w;
    n5 = in_n * loss_c * (loss_h+(loss_h-1)*(st_h-1)) * (loss_w+(loss_w-1)*(st_w-1));  // dilated loss tensor size  
    b1 = (float *)malloc (n5 * sizeof (float));  // dilate loss tensor area
    c = (float *)malloc (n3 * sizeof (float));

    //initialize c
    for(i=0;i<n3;i++){
        c[i] = 0.0;
    }
    //initialize b1
    for(i=0;i<n5;i++){
        b1[i] = 0.0;
    }
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b1, n5 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

    
    // copy from host a,b,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b1, b1, n5 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_n,1,1);
    gradfilter2_kernel <<<blocks, threads>> >(dev_a, dev_b1, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, loss_c, loss_h, loss_w, st_h, st_w, pad, in_c, in_h, in_w, in_n);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

    //average
    // clear d
    for(i=0;i<n4;i++){
        d[i] = 0.0;
    }
    // copy from c to d and compute sum
    for(i=0;i<in_n;i++){
        for(j=0;j<filt_n;j++){
            for(k=0;k<filt_c;k++){
                for(l=0;l<filt_h;l++){
                    for(m=0;m<filt_w;m++){
                        elt = c[IDX5C(i,j,k,l,m,filt_n,filt_c,filt_h,filt_w)];
                        d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] = d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] + elt;
                    }
                }
            }
        }
    }
    // average
    for(i=0;i<n4;i++){
        d[i] = d[i] / (float)in_n;
    }
     
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_b1);
    hipFree(dev_c);
    free(b1);
    free(c);
}



__global__ void full_kernel(float *a, float *b, int in_n, int in_c, int in_h, int in_w, int n)
{
    int tid = threadIdx.x;
    int n1,i,j,k;
    float elt;
    if(tid < n)
    {   
        n1 = tid;
        for(i=0;i<in_c;i++){
            for(j=0;j<in_h;j++){
                for(k=0;k<in_w;k++){
                    elt = a[IDX4C(n1,i,j,k,in_c,in_h,in_w)];
                    b[IDX2C(n1,i*in_h*in_w + j*in_w + k,in_n)] = elt;
                }
            }
        }
    }
}
  
/*
1st arg in_n of input tensor 4DIM
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg input tensor
6th arg output tensor
*/

void full1(int in_n, int in_c, int in_h, int in_w, float *a, float *b){
    int n1,n;
    float *dev_a, *dev_b;
 
    n1 = in_n * in_c * in_h * in_w;
    n = in_n;
      
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
  
    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

    full_kernel << <1, n>> >(dev_a, dev_b, in_n, in_c, in_h, in_w, n);
  
    // copy to host d from GPU dev_d
    CHECK(hipMemcpy(b, dev_b, n1 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);

}


__global__ void unfull_kernel(float *a, float *b, int in_n, int in_c, int in_h, int in_w, int n)
{
    int tid = threadIdx.x;
    int n1,i,j,k;
    float elt;
    if(tid < n)
    {   
        n1 = tid;
        for(i=0;i<in_c;i++){
            for(j=0;j<in_h;j++){
                for(k=0;k<in_w;k++){
                    elt = a[IDX2C(n1,i*in_h*in_w + j*in_w + k,in_n)];
                    b[IDX4C(n1,i,j,k,in_c,in_h,in_w)] = elt;
                }
            }
        }
    }
}
  
/*
1st arg in_n of input tensor 4DIM
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg input tensor
6th arg output tensor
*/

void unfull1(int in_n, int in_c, int in_h, int in_w, float *a, float *b){
    int n1,n;
    float *dev_a, *dev_b;
    
    n1 = in_n * in_c * in_h * in_w;
    n = in_n;
      
      // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
  
    // copy from host a,b1,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

    unfull_kernel << <1, n>> >(dev_a, dev_b, in_n, in_c, in_h, in_w, n);
  
    // copy to host d from GPU dev_d
    CHECK(hipMemcpy(b, dev_b, n1 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);

}

__global__ void emult1_kernel(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {
        c[tid] = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}


void emult1(int n, float *a, float *b,float *c) {
    float *dev_a, *dev_b, *dev_c;

    
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    emult1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}


void transpose1(int r1, int c1, float *a, float *b){
    int i, j;
    
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            b[IDX2C(j,i,c1)] = a[IDX2C(i,j,r1)];
        }
    }

}


void ident1(int n, float *a){
    int i,j;

    // Set matrix data 
    for(i=0;i<n;i++){
        for(j=0;j<n;j++){
            if(i==j)
                a[IDX2C(i,j,n)] = 1.0;
            else
                a[IDX2C(i,j,n)] = 0.0;
        }
    }

}


__global__ void sigmoid_kernel(float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        b[tid] = SIGMOID(a[tid]);
        tid += blockDim.x * gridDim.x;
    }
}

void activate_sigmoid(int n, float *a, float *b){
    float *dev_a, *dev_b;

   
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    sigmoid_kernel << <128, 128 >> >(dev_a, dev_b, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);

}

  
__global__ void tanh_kernel(float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {
        b[tid] = tanh(a[tid]);
        tid += blockDim.x * gridDim.x;
    }
}


void activate_tanh(int n, float *a, float *b){
    float *dev_a, *dev_b;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    tanh_kernel << <128, 128 >> >(dev_a, dev_b, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
}


  
__global__ void relu_kernel(float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        if(a[tid] >= 0)
            b[tid] = a[tid];
        else 
            b[tid] = 0.0;
        tid += blockDim.x * gridDim.x;
    }
}


void activate_relu(int n, float *a, float *b) {
    float *dev_a, *dev_b;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    relu_kernel << <128, 128 >> >(dev_a, dev_b, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
}

void activate_softmax(int r1, int c1, float *a, float *b){
    int i, j, k;
    float max,sum,delta;

    
    //calculate softmax
    delta = 0.01;
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            max = -3.402823e38;
            for(k=0;k<c1;k++){
                if(a[IDX2C(i,k,r1)] > max)
                    max = a[IDX2C(i,k,r1)];
            }
            sum = 0.0;
            for(k=0;k<c1;k++){
                sum = sum + exp(a[IDX2C(i,k,r1)] - max);
            }
            b[IDX2C(i,j,r1)] = exp(a[IDX2C(i,j,r1)] - max) / (sum+delta);
            
        }
    }

}



__global__ void differ_sigmoid_kernel(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        
        c[tid] = a[tid] * ((1 - SIGMOID(b[tid])) * SIGMOID(b[tid]));
        tid += blockDim.x * gridDim.x;
    }
}


void differ_sigmoid(int n, float *a, float *b, float *c){
    float *dev_a, *dev_b, *dev_c;

    
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    differ_sigmoid_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

}


__global__ void differ_tanh_kernel(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        c[tid] = a[tid] * (1/(cosh(b[tid]) * cosh(b[tid])));
        tid += blockDim.x * gridDim.x;
    }
}


void differ_tanh(int n, float *a, float *b, float *c){
    float *dev_a, *dev_b, *dev_c;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    differ_tanh_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}



__global__ void differ_relu_kernel(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        if(b[tid] >= 0)
            c[tid] = a[tid];
        else 
            c[tid] = 0.0;
        tid += blockDim.x * gridDim.x;
    }
}


void differ_relu(int n, float *a, float *b, float *c){
    float *dev_a, *dev_b, *dev_c;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    differ_relu_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

}



  
__global__ void smult_kernel(float d, float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {
        b[tid] = d * a[tid];
        tid += blockDim.x * gridDim.x;
    }
}



void smult1(float s, int n, float *a, float *b){
    float *dev_a, *dev_b;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    smult_kernel << <128, 128 >> >((float)s,dev_a, dev_b, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);

}

  
float trace1(int r1, int c1, float *a){
    int i, j;
    float trace;

    trace = 0.0;
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            if(i==j)
                trace = trace + a[IDX2C(i,j,r1)];
        }
    }

    return(trace);
}


float mean_square(int r1, int c1, float *a, float *b){
    int i, j;
    float d,s;

    
    s = 0.0;
    for(i=0;i<r1;i++){
        for (j=0;j<c1;j++){
            d = a[IDX2C(i,j,r1)] -  b[IDX2C(i,j,r1)];
            s = s + d*d;            
        }
    } 
    s = s / (2.0*(float(r1)));
    return(s);
}


float cross_entropy(int r1, int c1, float *a, float *b){
    int i, j;
    float d,s,delta;

    
    
    delta = 1e-7;
    s = 0.0;
    for(i=0;i<r1;i++){
        for (j=0;j<c1;j++){
            d = a[IDX2C(i,j,r1)] + delta;
            s = s + b[IDX2C(i,j,r1)] * log(d);
        }
    }
    s = -1.0 * s / (float)r1;
    return(s);
}



void add_diff1(int r1, int c1, float *a, float *b, int x, int y, float val) {
    int i, j;
    
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            if(i==x && j==y)
                b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)] + (float)val;
            else 
                b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)];
        }
    }

}


void add_diff2(int n1, int c1, int h1, int w1, float *a, float *b, int n2, int c2, int h2, int w2,float val){
    int i, j, k, l;
    
    for(i=0;i<n1;i++){
        for(j=0;j<c1;j++){
            for(k=0;k<h1;k++){
                for(l=0;l<w1;l++){
                    if(i==n2 && j==c2 && k==h2 && l==w2){
                        b[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(i,j,k,l,c1,h1,w1)] + (float)val;
                    }
                    else {
                        b[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(i,j,k,l,c1,h1,w1)];
                    }
                }
            }
        }
    }


}


void average1(int r1, int c1, float *a, float *b){
    int i, j;
    float sum;

    
    for(j=0;j<c1;j++){
        sum = 0.0;
        for(i=0;i<r1;i++){
            sum = sum + a[IDX2C(i,j,r1)];
        }
        b[j] = sum / (float)r1;
    }

}

/*
1st arg row-size of matrix
2nd arg col-size of matrix
3rd arg matrix data 
*/


float sum1(int r1, int c1, float *a, float *b){
    int i, j;
    float sum;

    
    sum = 0.0;
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            sum = sum + a[IDX2C(i,j,r1)];
        }
    }

    return(sum);
}


